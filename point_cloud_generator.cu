#include "hip/hip_runtime.h"
#include "point_cloud_generator.h"

__global__ void ComputeVertexMap(PtrStep<float3>output_vertex_map,
	                              const PtrStepSz<float> input_depth_map, 
	                              const float data_cutoff, 
	                              const CameraParams &camera_params)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.x*blockDim.y + threadIdx.y;

	if (x >= camera_params.image_width || y >= camera_params.image_height)
		return;

	float depth_value = input_depth_map.ptr(y)[x];
	if (depth_value > data_cutoff) depth_value = 0;

	float point_x = (x - camera_params.c_x)*depth_value / camera_params.focal_x;
	float point_y = (y - camera_params.c_y)*depth_value / camera_params.focal_y;

	output_vertex_map.ptr(y)[x] = make_float3(point_x, point_y, depth_value);

}