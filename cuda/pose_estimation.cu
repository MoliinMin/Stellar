#include "hip/hip_runtime.h"
#include "pose_estimation.h"

__global__ void PoseEstimate(const PtrStep<float> current_rotate_matrix, 
	                         const PtrStep<float> current_translate_matrix, 
	                         const PtrStep<float3> current_vertex_map, 
	                         const PtrStep<float3> current_normal_map, 
	                         const PtrStep<float> previous_rotate_matrix_inv, 
	                         const PtrStep<float> previous_translate_matrix, 
	                         const CameraParams camera_params, 
	                         const PtrStep<float3> previous_vertex_map, 
	                         const PtrStep<float3> previous_normal_map, 
	                         const float distance_threshold, 
	                         const float angle_threshold)
{

	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	































}
