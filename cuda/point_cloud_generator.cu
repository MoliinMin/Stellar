#include "hip/hip_runtime.h"
#include "point_cloud_generator.h"

__global__ void GpuComputeVertexMap(PtrStep<float3>output_vertex_map,
	                              const PtrStepSz<float> input_depth_map, 
	                              const float data_cutoff, 
	                              const CameraParams &camera_params)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.x*blockDim.y + threadIdx.y;

	if (x >= camera_params.image_width || y >= camera_params.image_height)
		return;

	float depth_value = input_depth_map.ptr(y)[x];
	if (depth_value > data_cutoff) depth_value = 0;

	float point_x = (x - camera_params.c_x)*depth_value / camera_params.focal_x;
	float point_y = (y - camera_params.c_y)*depth_value / camera_params.focal_y;

	output_vertex_map.ptr(y)[x] = make_float3(point_x, point_y, depth_value);

}

__global__ void GpuComputeNormalMap(PtrStep<float3> output_normal_map, const PtrStepSz<float3> input_vertex_map)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < 1 || x >= input_vertex_map.cols - 1 || y < 1 || y >= input_vertex_map.rows)
		return;

	float3 left_vertex = input_vertex_map.ptr(y)[x - 1];
	float3 right_vertex = input_vertex_map.ptr(y)[x + 1];
	float3 upper_vertex = input_vertex_map.ptr(y - 1)[x];
	float3 below_vertex = input_vertex_map.ptr(y + 1)[x];

	float3 normal;

	if (left_vertex.z == 0 || right_vertex.z == 0 || upper_vertex.z == 0 || below_vertex.z == 0)
	{
		normal = make_float3(0, 0, 0);
	}
	else
	{
		float3 hor = make_float3(left_vertex.x - right_vertex.x,
			left_vertex.y - right_vertex.y,
			left_vertex.z - right_vertex.z);
		float3 ver = make_float3(upper_vertex.x - below_vertex.x,
			upper_vertex.y - below_vertex.y,
			upper_vertex.z - below_vertex.z);

		Cross(hor, ver, normal);
		normalize(normal);
		if (normal.z > 0)
			normal = make_float3(-normal.x, normal.y, -normal.z);
	}
	output_normal_map.ptr(y)[x] = normal;
}

__device__ void Cross(const float3  target_vector, const float3 source_vector, float3 &output_cross_vector)
{

	float x = target_vector.y*source_vector.z - target_vector.z*source_vector.y;
	float y = -target_vector.x*source_vector.z + target_vector.z*source_vector.x;
	float z = target_vector.x*source_vector.y - target_vector.y*source_vector.x;
	output_cross_vector = make_float3(x, y, z);
}

__device__ void normalize(float3 &input_vector)
{

	float length=sqrt(input_vector.x*input_vector.x+ input_vector.y*input_vector.y+ input_vector.z*input_vector.z);
	float x = input_vector.x / length;
	float y = input_vector.y / length;
	float z = input_vector.z / length;
	input_vector = make_float3(x, y, z);
}

void ComputeVertexMap(GpuMat &output_vertex_map, const GpuMat &input_depth_map, const float data_cutoff, const CameraParams &camera_params)
{
	int cols = input_depth_map.cols;
	int rows = input_depth_map.rows;

	dim3 threads(32, 32);
	dim3 blocks((cols + threads.x - 1) / threads.x, (rows + threads.y - 1) / threads.y);

	GpuComputeVertexMap << <blocks, threads >> > (output_vertex_map, input_depth_map, data_cutoff, camera_params);
	hipDeviceSynchronize();
}


void ComputeNormalMap(GpuMat &output_normal_map, const GpuMat &input_vertex_map)
{
	int cols = input_vertex_map.cols;
	int rows = input_vertex_map.rows;

	dim3 threads(32, 32);
	dim3 blocks((cols + threads.x - 1) / threads.x, (rows + threads.y - 1) / threads.y);

	GpuComputeNormalMap<<<blocks,threads>>>(output_normal_map, input_vertex_map);
	hipDeviceSynchronize(); 
}